#include "hip/hip_runtime.h"
#include <iostream>
#include "device_functions.cuh"

__global__ void compute(float *d_result) {
    *d_result = multiply(3.0f, 4.0f);
}

int main() {
    float *d_result, h_result;
    hipMalloc(&d_result, sizeof(float));
    
    compute<<<1, 1>>>(d_result);
    hipMemcpy(&h_result, d_result, sizeof(float), hipMemcpyDeviceToHost);
    
    std::cout << "Result: " << h_result << std::endl;
    hipFree(d_result);
    return 0;
}

